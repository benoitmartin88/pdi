#include "hip/hip_runtime.h"
#include <stdio.h>
#include "wrapperFunc.h"


#define CUDA_CALL_SAFE(f)                                                                       \
  do {                                                                                            \
    hipError_t _e = f;                                                                          \
    if(_e != hipSuccess) {                                                                    \
      fprintf(stderr, "Cuda error %s %d %s:: %s\n", __FILE__,__LINE__, __func__, hipGetErrorString(_e));  \
      exit(EXIT_FAILURE);                                                                       \
    }                                                                                           \
  } while(0)

__global__
void vecmultGPU(double* A, double* B, size_t asize) {
  size_t i = (size_t)blockIdx.x * (size_t)blockDim.x + (size_t)threadIdx.x;
  if (i < asize) {
    A[i] = A[i] * B[i];
  }
}


void allocateMemory(void **ptr, size_t size){
  CUDA_CALL_SAFE(hipMalloc(ptr, size));
  return;
}

void cudaCopy(void *src, void *dest, size_t size){
  CUDA_CALL_SAFE(hipMemcpy(dest, src, size, hipMemcpyHostToDevice));
}

void executeVecMult(int blocks, int threads, double *A, double *B, size_t size, double *ha){
  vecmultGPU<<< blocks, threads >>>(A, B, size);
  CUDA_CALL_SAFE(hipDeviceSynchronize());
  CUDA_CALL_SAFE(hipMemcpy(ha, A, size*sizeof(double) , hipMemcpyDeviceToHost));
}

void freeCuda( void *ptr ){
  CUDA_CALL_SAFE(hipFree(ptr));
}


void executeVecMultUnified(int blocks, int threads, double *A, double *B, size_t size, double *ha){
  vecmultGPU<<< blocks, threads >>>(A, B, size);
  CUDA_CALL_SAFE(hipDeviceSynchronize());
}


void allocateManaged( void **ptr , size_t size){
  CUDA_CALL_SAFE(hipMallocManaged(ptr, size));
}

int getProperties(){
  int nDevices;
  hipGetDeviceCount(&nDevices);
  return nDevices;
}

void setDevice(int id){
  CUDA_CALL_SAFE(hipSetDevice(id));
}
