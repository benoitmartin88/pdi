#include "hip/hip_runtime.h"
#include "wrapperFunc.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define CUDA_CALL_SAFE(f)                                                                       \
  do {                                                                                            \
    hipError_t _e = f;                                                                          \
    if(_e != hipSuccess) {                                                                    \
      fprintf(stderr, "Cuda error %s %d %s:: %s\n", __FILE__,__LINE__, __func__, hipGetErrorString(_e));  \
      exit(EXIT_FAILURE);                                                                       \
    }                                                                                           \
  } while(0)

__global__ void mykernel(int* data, int start){
  int myId = blockIdx.x * blockDim.x +  threadIdx.x;
  data[myId] = start+myId;
}

void allocateMemory(void **ptr, size_t size){
  CUDA_CALL_SAFE(hipMalloc(ptr, size));
  return;
}

void cudaCopy(void *src, void *dest, size_t size){
  CUDA_CALL_SAFE(hipMemcpy(dest, src, size, hipMemcpyHostToDevice));
}

void hostCopy(void *src, void *dest, size_t size){
  CUDA_CALL_SAFE(hipMemcpy(dest, src, size, hipMemcpyDeviceToHost));
}

void freeCuda( void *ptr ){
  CUDA_CALL_SAFE(hipFree(ptr));
}

void deviceMemset(void *ptr, int size){
  CUDA_CALL_SAFE( hipMemset(ptr, 0, size) );
}

void executeKernel( int *ptr, int start){
  mykernel<<<1024, 1024>>>(ptr, start);
  CUDA_CALL_SAFE(hipPeekAtLastError());
  CUDA_CALL_SAFE(hipDeviceSynchronize());
}

int getProperties(){
  int nDevices;
  hipGetDeviceCount(&nDevices);
  return nDevices;
}

void setDevice(int id){
  CUDA_CALL_SAFE(hipSetDevice(id));
}
